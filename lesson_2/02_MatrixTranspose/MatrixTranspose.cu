#include "hip/hip_runtime.h"
#include <chrono>
#include <iomanip>
#include <iostream>
#include <random>
#include "Timer.cuh"
#include "CheckError.cuh"
using namespace timer;

const int BLOCK_SIZE_X = 16;
const int BLOCK_SIZE_Y = 16;

__global__
void matrixTransposeKernel(const int* d_matrix_in,
                           int        N,
                           int*       d_matrix_out) {
    __shared__ int ds_matrix_in[BLOCK_SIZE_X][BLOCK_SIZE_Y];

    int row = blockIdx.y * BLOCK_SIZE_Y + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE_X + threadIdx.x;

    if(row < N && col < N) {
        ds_matrix_in[threadIdx.y][threadIdx.x] = d_matrix_in[row*N + col];
        __syncthreads();

        row = blockIdx.x * blockDim.x + threadIdx.y;
        col = blockIdx.y * blockDim.y + threadIdx.x;

        d_matrix_out[row*N + col] = ds_matrix_in[threadIdx.x][threadIdx.y];
    }
}

const int N  = 1000;

int main() {
    Timer<DEVICE> TM_device;
    Timer<HOST>   TM_host;
    // -------------------------------------------------------------------------
    // HOST MEMORY ALLOCATION
    int* h_matrix_in  = new int[N * N];
    int* h_matrix_tmp = new int[N * N]; // <-- used for device result
    int* h_matrix_out = new int[N * N];

    // -------------------------------------------------------------------------
    // HOST INITILIZATION
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator(seed);
    std::uniform_int_distribution<int> distribution(1, 100);

    for (int i = 0; i < N * N; i++)
        h_matrix_in[i] = distribution(generator);

    // -------------------------------------------------------------------------
    // HOST EXECUTIION
    TM_host.start();

    for (int i = 0; i < N ; i++) {
        for (int j = 0; j < N ; j++)
            h_matrix_out[i * N + j] = h_matrix_in[j * N + i];
    }

    TM_host.stop();
    TM_host.print("MatrixTranspose host:   ");

    // -------------------------------------------------------------------------
    // DEVICE MEMORY ALLOCATION
    int *d_matrix_in, *d_matrix_out;
    SAFE_CALL( hipMalloc( &d_matrix_in, N*N * sizeof(int) ) );
    SAFE_CALL( hipMalloc( &d_matrix_out, N*N * sizeof(int) ) );

    // -------------------------------------------------------------------------
    // COPY DATA FROM HOST TO DEVIE
    SAFE_CALL( hipMemcpy( d_matrix_in, h_matrix_in, N*N * sizeof(int), hipMemcpyHostToDevice ) );

    // -------------------------------------------------------------------------
    // DEVICE EXECUTION
    TM_device.start();

    dim3 num_blocks(N/BLOCK_SIZE_X, N/BLOCK_SIZE_Y, 1);
    if (N % BLOCK_SIZE_X) num_blocks.x++;
    if (N % BLOCK_SIZE_Y) num_blocks.y++;
    dim3 dim_block(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
    matrixTransposeKernel<<< num_blocks, dim_block >>>(d_matrix_in, N, d_matrix_out);

    TM_device.stop();
    CHECK_CUDA_ERROR
    TM_device.print("MatrixTranspose device: ");

    std::cout << std::setprecision(1)
              << "Speedup: " << TM_host.duration() / TM_device.duration()
              << "x\n\n";

    // -------------------------------------------------------------------------
    // COPY DATA FROM DEVICE TO HOST
    SAFE_CALL( hipMemcpy( h_matrix_tmp, d_matrix_out, N*N * sizeof(int), hipMemcpyDeviceToHost ) );


    // -------------------------------------------------------------------------
    // RESULT CHECK
    for (int i = 0; i < N * N; i++) {
        if (h_matrix_out[i] != h_matrix_tmp[i]) {
            std::cerr << "wrong result at: ("
                      << (i / N) << ", " << (i % N) << ")"
                      << "\nhost:   " << h_matrix_out[i]
                      << "\ndevice: " << h_matrix_tmp[i] << "\n\n";
            hipDeviceReset();
            std::exit(EXIT_FAILURE);
        }
    }
    std::cout << "<> Correct\n\n";

    // -------------------------------------------------------------------------
    // HOST MEMORY DEALLOCATION
    delete[] h_matrix_in;
    delete[] h_matrix_out;
    delete[] h_matrix_tmp;

    // -------------------------------------------------------------------------
    // DEVICE MEMORY DEALLOCATION
    SAFE_CALL( hipFree( d_matrix_in ) );
    SAFE_CALL( hipFree( d_matrix_out ) );

    // -------------------------------------------------------------------------
    hipDeviceReset();
}
