#include "hip/hip_runtime.h"
#include <chrono>
#include <iomanip>
#include <iostream>
#include <random>
#include "Timer.cuh"
#include "CheckError.cuh"
using namespace timer;

__global__
void matrixTransposeKernel(const int* d_matrix_in,
                           int        N,
                           int*       d_matrix_out) {
    int row = blockIdx.y+blockDim.y + threadIdx.y;
    int col = blockIdx.x+blockDim.x + threadIdx.x;

    d_matrix_out[Row * N + Col] = d_matrix_in[Col * N + Row];
}

const int N  = 1024;
const int BLOCK_SIZE_X = 16;
const int BLOCK_SIZE_Y = 16;

int main() {
    Timer<DEVICE> TM_device;
    Timer<HOST>   TM_host;
    // -------------------------------------------------------------------------
    // HOST MEMORY ALLOCATION
    int* h_matrix_in  = new int[N * N];
    int* h_matrix_tmp = new int[N * N]; // <-- used for device result
    int* h_matrix_out = new int[N * N];

    // -------------------------------------------------------------------------
    // HOST INITILIZATION
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator(seed);
    std::uniform_int_distribution<int> distribution(1, 100);

    for (int i = 0; i < N * N; i++)
        h_matrix_in[i] = distribution(generator);

    // -------------------------------------------------------------------------
    // HOST EXECUTIION
    TM_host.start();

    for (int i = 0; i < N ; i++) {
        for (int j = 0; j < N ; j++)
            h_matrix_out[i * N + j] = h_matrix_in[j * N + i];
    }

    TM_host.stop();
    TM_host.print("MatrixTranspose host:   ");

    // -------------------------------------------------------------------------
    // DEVICE MEMORY ALLOCATION
    int *d_matrix_in, *d_matrix_out;
    SAFE_CALL( hipMalloc( &d_matrix_in, N*N*sizeof(int) ) )
    SAFE_CALL( hipMalloc( &d_matrix_out, N*N*sizeof(int)) )

    // -------------------------------------------------------------------------
    // COPY DATA FROM HOST TO DEVIE
    SAFE_CALL( hipMemcpy( d_matrix_in, h_matrix_in, N * N * sizeof(int), hipMemcpyHostToDevice));
    // -------------------------------------------------------------------------
    // DEVICE EXECUTION
    TM_device.start();

    dim3 DimGrid(N/BLOCK_SIZE_X, N/BLOCK_SIZE_Y, 1);
    if (N%BLOCK_SIZE_X) DimGrid.x++;
    if (N%BLOCK_SIZE_Y) DimGrid.y++;
    dim3 DimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
    matrixTransposeKernel<<<DimGrid,DimBlock>>>(d_matrix_in, N, d_matrix_out);

    TM_device.stop();
    CHECK_CUDA_ERROR
    TM_device.print("MatrixTranspose device: ");

    std::cout << std::setprecision(1)
              << "Speedup: " << TM_host.duration() / TM_device.duration()
              << "x\n\n";

    // -------------------------------------------------------------------------
    // COPY DATA FROM DEVICE TO HOST
    SAFE_CALL( hipMemcpy( h_matrix_tmp, d_matrix_out, N*N*sizeof(int), hipMemcpyDeviceToHost) )

    // -------------------------------------------------------------------------
    // RESULT CHECK
    for (int i = 0; i < N * N; i++) {
        if (h_matrix_out[i] != h_matrix_tmp[i]) {
            std::cerr << "wrong result at: ("
                      << (i / N) << ", " << (i % N) << ")"
                      << "\nhost:   " << h_matrix_out[i]
                      << "\ndevice: " << h_matrix_tmp[i] << "\n\n";
            hipDeviceReset();
            std::exit(EXIT_FAILURE);
        }
    }
    std::cout << "<> Correct\n\n";

    // -------------------------------------------------------------------------
    // HOST MEMORY DEALLOCATION
    delete[] h_matrix_in;
    delete[] h_matrix_out;
    delete[] h_matrix_tmp;

    // -------------------------------------------------------------------------
    // DEVICE MEMORY DEALLOCATION
    SAFE_CALL( hipFree( d_matrix_in ) )
    SAFE_CALL( hipFree( d_matrix_out ) )

    // -------------------------------------------------------------------------
    hipDeviceReset();
}
