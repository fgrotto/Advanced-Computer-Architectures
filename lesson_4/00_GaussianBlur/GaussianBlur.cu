#include "hip/hip_runtime.h"
#include <chrono>
#include <math.h>
#include <iomanip>
#include <iostream>
#include "Timer.cuh"
#include "CheckError.cuh"
#include <opencv2/opencv.hpp>
using namespace timer;

const int N = 10; // it has to be divisible by 2
const int WIDTH  =1000;
const int HEIGHT = 500;
const int CHANNELS = 3;
const int BLOCK_SIZE = 32;

template <class T>
void printImageForDebug(T *image, int N, int width, int channels)
{
	std::cout << std::endl;
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			std::cout << "[ ";
			for (int k = 0; k < channels; k++)
			{
				std::cout << image[(i * width + j) * channels + k];
				k == channels - 1 ? std::cout << " " : std::cout << " , ";
			}
			std::cout << "]";
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;
}


void buildGaussianFilterMask(float *Mask, int N, float sigma)
{	
    double r, s = 2.0 * sigma * sigma;
    double sum = 0.0;

    for (int x = -N/2; x < N/2; x++) {
        for (int y = -N/2; y < N/2; y++) {
            r = sqrt(x * x + y * y);
            Mask[(x + N/2) * N + (y + N/2)] = (exp(-(r * r) / s)) / (M_PI * s);
            sum += Mask[(x + N/2) * N + (y + N/2)];
        }
    }
 
    for (int i = 0; i < N; ++i)
        for (int j = 0; j < N; ++j)
            Mask[i *N + j] /= sum;
}

__global__ void GaussianBlurDevice(const unsigned char *image,
								   const float *mask,
								   unsigned char *image_out,
								   int N)
{

	int globalId_x = threadIdx.x + blockIdx.x * blockDim.x;
	int globalId_y = threadIdx.y + blockIdx.y * blockDim.y;

	if (globalId_x < WIDTH && globalId_y < HEIGHT)
	{
		for (int channel = 0; channel < CHANNELS; channel++)
		{
			float pixel_value = 0;
			for (int u = 0; u < N; u++)
			{
				for (int v = 0; v < N; v++)
				{
					int new_x = min(WIDTH, max(0, globalId_x + u - N / 2));
					int new_y = min(HEIGHT, max(0, globalId_y + v - N / 2));
					pixel_value += mask[v * N + u] * image[(new_y * WIDTH + new_x) * CHANNELS + channel];
				}
			}
			image_out[(globalId_y * WIDTH + globalId_x) * CHANNELS + channel] = (unsigned char)pixel_value;
		}
	}
}

void GaussianBlurHost(const unsigned char *image,
					  const float *mask,
					  unsigned char *image_out)
{

	for (int y = 0; y < HEIGHT; y++)
	{
		for (int x = 0; x < WIDTH; x++)
		{
			for (int channel = 0; channel < CHANNELS; channel++)
			{
				float pixel_value = 0;
				for (int u = 0; u < N; u++)
				{
					for (int v = 0; v < N; v++)
					{
						int new_x = min(WIDTH, max(0, x + u - N / 2));
						int new_y = min(HEIGHT, max(0, y + v - N / 2));
						pixel_value += mask[v * N + u] * image[(new_y * WIDTH + new_x) * CHANNELS + channel];
					}
				}
				image_out[(y * WIDTH + x) * CHANNELS + channel] = (unsigned char)pixel_value;
			}
		}
	}
}

int main()
{
	Timer<DEVICE> TM_device;
	Timer<HOST> TM_host;

	cv::Mat img = cv::imread("../image.png");

	if (img.empty())
	{
		std::cout << "Failed imread(): image not found" << std::endl;
		exit(0);
	}

	// cv::namedWindow("Display window");
	// cv::imshow("Display window", img);
	// cv::waitKey(0);

	// -------------------------------------------------------------------------
	// HOST MEMORY ALLOCATION
	unsigned char *image = new unsigned char[WIDTH * HEIGHT * CHANNELS];
	unsigned char *host_image_out = new unsigned char[WIDTH * HEIGHT * CHANNELS];
	unsigned char *device_image_out = new unsigned char[WIDTH * HEIGHT * CHANNELS];

	float *mask = new float[N * N];
	float sigma = 1.0;
	image = img.data;

	// Build a gaussian filter for the image
	buildGaussianFilterMask(mask, N, sigma);

	// -------------------------------------------------------------------------
	// HOST EXECUTIION
	TM_host.start();

	GaussianBlurHost(image, mask, host_image_out);

	TM_host.stop();
	TM_host.print("GaussianBlur host:   ");

	// cv::Mat A(HEIGHT, WIDTH, CV_8UC3, host_image_out);
	// cv::imshow("Result of gaussian blur (host)", A);
	// cv::waitKey(0);

	// -------------------------------------------------------------------------
	// DEVICE MEMORY ALLOCATION

	unsigned char *dev_image, *dev_image_out;
	float *dev_mask;

	SAFE_CALL(hipMalloc(&dev_image, WIDTH * HEIGHT * CHANNELS * sizeof(unsigned char)));
	SAFE_CALL(hipMalloc(&dev_image_out, WIDTH * HEIGHT * CHANNELS * sizeof(unsigned char)));
	SAFE_CALL(hipMalloc(&dev_mask, N * N * sizeof(float)));

	// -------------------------------------------------------------------------
	// COPY DATA FROM HOST TO DEVICE

	SAFE_CALL(hipMemcpy(dev_image, image, WIDTH * HEIGHT * CHANNELS * sizeof(unsigned char), hipMemcpyHostToDevice));
	SAFE_CALL(hipMemcpy(dev_mask, mask, N * N * sizeof(float), hipMemcpyHostToDevice));

	// -------------------------------------------------------------------------
	// DEVICE EXECUTION

	dim3 block_size(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 num_blocks(ceil(float(WIDTH) / BLOCK_SIZE), ceil(float(HEIGHT) / BLOCK_SIZE), 1);

	TM_device.start();

	GaussianBlurDevice<<<block_size, num_blocks>>>(dev_image, dev_mask, dev_image_out, N);

	TM_device.stop();
	CHECK_CUDA_ERROR
	TM_device.print("GaussianBlur device: ");

	std::cout << std::setprecision(1)
			  << "Speedup: " << TM_host.duration() / TM_device.duration()
			  << "x\n\n";

	// -------------------------------------------------------------------------
	// COPY DATA FROM DEVICE TO HOST

	SAFE_CALL(hipMemcpy(device_image_out, dev_image_out, WIDTH * HEIGHT * CHANNELS * sizeof(unsigned char), hipMemcpyDeviceToHost));

	// -------------------------------------------------------------------------
	// RESULT CHECK

	// cv::Mat B(HEIGHT, WIDTH, CV_8UC3, device_image_out);
	// cv::imshow("Result of gaussian blur (device)", B);
	// cv::waitKey(0);

	for (int i = 0; i < HEIGHT; i++)
	{
		for (int j = 0; j < WIDTH; j++)
		{
			if (device_image_out[i * WIDTH + j] != host_image_out[i * WIDTH + j] 
					|| device_image_out[i * WIDTH + j] != host_image_out[i * WIDTH + j])
			{
				std::cerr << "wrong result at [" << i << "][" << j << "]!" << std::endl;
				std::cerr << "image_out: " << (short)host_image_out[i * WIDTH + j] << std::endl;
				std::cerr << "device_image_out: " << (short)device_image_out[i * WIDTH + j] << std::endl;
				hipDeviceReset();
				std::exit(EXIT_FAILURE);
			}
		}
	}
	std::cout << "<> Correct\n\n";

	// -------------------------------------------------------------------------
	// HOST MEMORY DEALLOCATION
	delete[] host_image_out;
	delete[] device_image_out;
	delete[] mask;

	// -------------------------------------------------------------------------
	// DEVICE MEMORY DEALLOCATION
	SAFE_CALL(hipFree(dev_image))
	SAFE_CALL(hipFree(dev_image_out))
	SAFE_CALL(hipFree(dev_mask))

	// -------------------------------------------------------------------------
	//SAFE_CALL(hipFree());
	hipDeviceReset();
}
