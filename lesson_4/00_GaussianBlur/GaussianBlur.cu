#include <chrono>
#include <iomanip>
#include <iostream>
#include "Timer.cuh"
#include "CheckError.cuh"
using namespace timer;

int main() {
    Timer<DEVICE> TM_device;
    Timer<HOST>   TM_host;
    
    // -------------------------------------------------------------------------
    // READ INPUT IMAGE


    // -------------------------------------------------------------------------
    // HOST MEMORY ALLOCATION


    // -------------------------------------------------------------------------
    // HOST EXECUTIION
    TM_host.start();

    /// GaussianBlurHost();

    TM_host.stop();
    TM_host.print("GaussianBlur host:   ");

    // -------------------------------------------------------------------------
    // DEVICE MEMORY ALLOCATION
    

    // -------------------------------------------------------------------------
    // COPY DATA FROM HOST TO DEVIE
    

    // -------------------------------------------------------------------------
    // DEVICE EXECUTION
    TM_device.start();

    /// GaussianBlurDevice<<<  >>>();

    TM_device.stop();
    CHECK_CUDA_ERROR
    TM_device.print("GaussianBlur device: ");

    std::cout << std::setprecision(1)
              << "Speedup: " << TM_host.duration() / TM_device.duration()
              << "x\n\n";

    // -------------------------------------------------------------------------
    // COPY DATA FROM DEVICE TO HOST


    // -------------------------------------------------------------------------
    // RESULT CHECK
    if (true /* Correct check here */) {
        std::cerr << "wrong result!" << std::endl;
        hipDeviceReset();
        std::exit(EXIT_FAILURE);
    }
    std::cout << "<> Correct\n\n";

    // -------------------------------------------------------------------------
    // HOST MEMORY DEALLOCATION


    // -------------------------------------------------------------------------
    // DEVICE MEMORY DEALLOCATION


    // -------------------------------------------------------------------------
    hipDeviceReset();
}
